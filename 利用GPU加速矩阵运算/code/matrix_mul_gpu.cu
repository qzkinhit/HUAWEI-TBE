#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <time.h> 
#include <stdio.h>
#include <math.h>

const int Row = 1024;
const int Col = 1024;

__global__
void matrix_mul_gpu(float* M, float* N, float* P, int width)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = threadIdx.y + blockDim.y * blockIdx.y;

    float sum = 0.0;
    for (int k = 0; k < width; k++)
    {
        float a = M[j * width + k];
        float b = N[k * width + i];
        sum += a * b;
    }
    P[j * width + i] = sum;
}

int main()
{
 
    clock_t start, end;
    start= clock();
    float* A = (float*)malloc(sizeof(float) * Row * Col);
    float* B = (float*)malloc(sizeof(float) * Row * Col);
    float* C = (float*)malloc(sizeof(float) * Row * Col);
    //malloc device memory
    float* d_dataA, * d_dataB, * d_dataC;
    hipMalloc((void**)&d_dataA, sizeof(float) * Row * Col);
    hipMalloc((void**)&d_dataB, sizeof(float) * Row * Col);
    hipMalloc((void**)&d_dataC, sizeof(float) * Row * Col);
    //set value
    for (int i = 0; i < Row * Col; i++) {
        A[i] = 19.0;
        B[i] = 20.0;
    }

    hipMemcpy(d_dataA, A, sizeof(float) * Row * Col, hipMemcpyHostToDevice);
    hipMemcpy(d_dataB, B, sizeof(float) * Row * Col, hipMemcpyHostToDevice);
    dim3 threadPerBlock(14, 14);
    dim3 blockNumber((Col + threadPerBlock.x - 1) / threadPerBlock.x, (Row + threadPerBlock.y - 1) / threadPerBlock.y);
    printf("Block(%d,%d)   Grid(%d,%d).\n", threadPerBlock.x, threadPerBlock.y, blockNumber.x, blockNumber.y);
    matrix_mul_gpu << <blockNumber, threadPerBlock >> > (d_dataA, d_dataB, d_dataC, Col);
    //������������-һ������ָ��
    hipMemcpy(C, d_dataC, sizeof(float) * Row * Col, hipMemcpyDeviceToHost);

    //�ͷ��ڴ�
    free(A);
    free(B);
    free(C);
    hipFree(d_dataA);
    hipFree(d_dataB);
    hipFree(d_dataC);
    end = clock();
    int timeuse =(end - start);
    printf("total time is %d ms\n", timeuse);

    return 0;
}